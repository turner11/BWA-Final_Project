
#include "hip/hip_runtime.h"
#include ""
#include "backend.h"
#include "utils.h"
#include "Source.h"

#include <stdio.h>



int main()
{
	printf("Hello Avis\n");
	
	//---------------------------
	int seqsCount = 0;
	bwa_seq_t* seqs = GetAllSequencies(&seqsCount);	
	//----------------------------------
	bwt_t *bwt = GetBwt();
	//----------------------------------
	

	int tid = 0;	
	int n_seqs =seqsCount;//41; 
	gap_opt_t *opt = (gap_opt_t *)malloc(sizeof(gap_opt_t ));
	opt->s_mm = 3;
	opt->s_gapo = 11;
	opt->s_gape = 4;
	opt->mode = 3;
	opt->indel_end_skip = 5;
	opt->max_del_occ = 10;
	opt->max_entries = 2000000;
	opt->fnr = 0.39999991;
	opt->max_diff = -1;
	opt->max_gapo = 1;
	opt->max_gape = 6;
	opt->max_seed_diff = 2;
	opt->seed_len=32;
	opt->n_threads = 1;
	opt->max_top2 = 30;
	opt->trim_qual = 0;


	bwa_cal_sa_reg_gapWithCuda(tid, bwt, n_seqs, seqs, opt);
	

	return 0;
}
